#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//function declaration
unsigned int getmax(unsigned int *, unsigned int);
//unsigned int getmaxSeq(unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2) {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }

    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers ) {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1
    for( i = 0; i < size; i++) {
      numbers[i] = rand()  % size;
    }
    printf("The maximum number in the array is: %u\n", getmax(numbers, size));

    free(numbers);
    exit(0);
}

__global__ void getmaxcu(unsigned int* num, int size, int threadCount)
{
  __shared__ int localBiggest[32];
  if (threadIdx.x==0) {
    for (int i = 0; i < 32; i++) {
      localBiggest[i] = 0;
    }
  }
  __syncthreads();

	int current =  blockIdx.x *blockDim.x + threadIdx.x;   //get current thread ID
  int localBiggestCurrent = (current - blockIdx.x *blockDim.x)/32;   //get currentID's warp number
  //if current number is bigger than the biggest number so far in the warp, replace it
  if ((num[current] > localBiggest[localBiggestCurrent]) && (current < size)) {
    localBiggest[localBiggestCurrent] = num[current];
  }
  __syncthreads();

  //using only one thread, loop through all the biggest numbers in each warp
  //and return the biggest number out of them all
  if (threadIdx.x==0) {
    int biggest = localBiggest[0];
    for (int i = 1; i < 32; i++) {
      if (biggest < localBiggest[i]) {
        biggest = localBiggest[i];
      }
    }
    //once found the biggest number in this block, put back into global array
    //num with corresponding block number
    num[blockIdx.x] = biggest;
  }

}

unsigned int getmax(unsigned int num[], unsigned int size)
{
  //get max threads per block. Since the two devices on the GPU cluster are the same,
  //I only got the property from one of the device
  int maxThreadsPerBlock, block;
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  maxThreadsPerBlock = prop.maxThreadsPerBlock;
  //get numbers of blocks needed depending on size and max threads per block
  block = (size / maxThreadsPerBlock) + 1;
  if (size % maxThreadsPerBlock == 0) {
    block = size / maxThreadsPerBlock;
  }

	unsigned int* device_num;
	hipSetDevice(1);
	hipMalloc((void **) &device_num, size*sizeof(unsigned int));

	hipMemcpy(device_num, num, size*sizeof(unsigned int), hipMemcpyHostToDevice);
	getmaxcu<<<block,maxThreadsPerBlock, 32>>>(device_num, size, maxThreadsPerBlock);
	hipMemcpy(num, device_num, size*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(device_num);

  //using what we calculated, get the biggest number from each block
  int answer = num[0];
  for (int i = 1; i < block; i++) {
    if (answer < num[i]) {
      answer = num[i];
    }
  }
  return answer;
}
